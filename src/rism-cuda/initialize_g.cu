#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include "rism3d.h"

void RISM3D :: initialize_g() {
  __global__ void set_g(double4 * dgv, double * dg2, 
			double bx, double by, double bz,
			int nx, int ny, int nz);

  indga = new int[ce -> ngrid];
  double * g2 = new double[ce -> ngrid];
  int * indg2 = new int[ce -> ngrid];

  double * dg2;
  hipMalloc(&dgv, ce -> ngrid * sizeof(double4));
  hipMalloc(&dg2, ce -> ngrid * sizeof(double));

  set_g <<< g, b >>> (dgv, dg2, ce -> box[0], ce -> box[1], ce -> box[2],
		      ce -> grid[0], ce -> grid[1], ce -> grid[2]);
  
  hipMemcpyAsync(g2, dg2, ce -> ngrid * sizeof(double), hipMemcpyDefault);
  thrust::device_vector<int> indg(ce -> ngrid);
  thrust::device_ptr<double> dg2_ptr(dg2);
  thrust::sequence(indg.begin(), indg.end());
  thrust::sort_by_key(dg2_ptr, dg2_ptr + ce -> ngrid, indg.begin());  
  thrust::copy(indg.begin(), indg.end(), indg2);

  double ga2o = - 1.0;
  nga = 0;

  for (int igk = 0; igk < ce -> ngrid; ++igk) {
    int igs = indg2[igk];
    double ga2 = g2[igs];
    if (ga2 > ga2o) {
      ++nga;
      ga . push_back (sqrt(ga2));
      ga2o = ga2;
    }
    indga[igs] = nga - 1;
  }

  hipFree(dg2);
  delete[] g2;
  delete[] indg2;
}


__global__ void set_g(double4 * dgv, double * dg2, 
		      double bx, double by, double bz, 
		      int nx, int ny, int nz) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  dgv[ip].x = 2.0 * M_PI * (threadIdx.x - nx / 2.0 + 0.5) / bx;
  dgv[ip].y = 2.0 * M_PI * (blockIdx.x - ny / 2.0 + 0.5) / by;
  dgv[ip].z = 2.0 * M_PI * (blockIdx.y - nz / 2.0 + 0.5) / bz;
  dg2[ip] = dgv[ip].x * dgv[ip].x + dgv[ip].y * dgv[ip].y 
    + dgv[ip].z * dgv[ip].z;
}
