#include <iostream>
#include <fstream>
#include <unistd.h>
#include "rism3d.h"

int main (int argc, char * argv[]) {
  RISM3D * system;
  int ch;
  int cu, dn;
  string input;
  string structure;
  bool centering = true;

  cu = dn = 0;
  system = new RISM3D;

  while ((ch = getopt(argc, argv, "c:d:i:s:f")) != -1) {
    switch (ch){
    case 'c':
      cu = atoi(optarg);
      break;
    case 'd':
      dn = atoi(optarg);
      break;
    case 'i':
      input = optarg;
      break;
    case 's':
      structure = optarg;
      break;
    case 'f':
      centering = false;
      break;
    }
  }

  if (input.empty() || structure.empty()) {
    if (argv[optind] == NULL) {
      cout << "No input file!" << endl;
      return (1);
    }
    input = argv[optind];
  }

  cout << "Set device " << dn << endl ;
  hipSetDevice(dn);
  if (cu > 0) cout << "Charge up " << cu << endl;
  system -> initialize(input, structure, centering);
  system -> iterate(cu);
  system -> output();    

  return(0);
}
