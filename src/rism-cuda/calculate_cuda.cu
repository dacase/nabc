
#include <hip/hip_runtime.h>
__global__ void kh(double * dtr, const double * __restrict__ dt, 
		   const double * __restrict__ du, const double * __restrict__ de, 
		   double q) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  double earg = - du[ip] - de[ip] * q + dt[ip];
  if (earg >= 0.0) {
    dtr[ip] = 1.0 + earg;
  } else {
    dtr[ip] = exp(earg);
  }
}

__global__ void hnc(double * dtr, const double * __restrict__ dt, 
		    const double * __restrict__ du, const double * __restrict__ de,
		    double q) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  dtr[ip] = exp(- du[ip] - de[ip] * q + dt[ip]);
}

__global__ void trm1mt(double2 * dguv, const double * __restrict__ dtr, 
		       const double * __restrict__ dt, 
		       const double * __restrict__ dfr, double qv) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  dguv[ip].x = dtr[ip] - 1.0 - dt[ip] + qv * dfr[ip];
  dguv[ip].y = 0.0;
}

__global__ void pqvfr(double2 * dguv, double * dfr, double qv) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  dguv[ip].x += qv * dfr[ip];
}

__global__ void mqvfk(double2 * dguv, const double2 * __restrict__ dfk, double qv) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  dguv[ip].x -= qv * dfk[ip].x;
  dguv[ip].y -= qv * dfk[ip].y;
}

__global__ void oz(double2 * dhuv, const double2 * __restrict__ dguv, 
		   const double * __restrict__ dx, int natv) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  unsigned int ngr = blockDim.x * gridDim.x * gridDim.y;
  double hr = 0.0;
  double hi = 0.0;
  for (unsigned int iv = 0; iv < natv; ++iv) {
    unsigned int i = ip + iv * ngr;
    hr += dguv[i].x * dx[i];
    hi += dguv[i].y * dx[i];
  }
  dhuv[ip].x = hr;
  dhuv[ip].y = hi;
}

__global__ void tr(double2 * dguv, double * dtr, const double2 * __restrict__ dhuv) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  dguv[ip].x = dtr[ip];
  dtr[ip] = dhuv[ip].x + 1.0 - dguv[ip].x;
  //  dguv[ip].y = 0.0;
}
