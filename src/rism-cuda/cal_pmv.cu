#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include "rism3d.h"

double RISM3D :: cal_pmv () {
  __global__ void pmv_cuv(double * ds, double2 * dhuv, double * dt);

  double cuv = 0.0;
  for (int iv = 0; iv < sv -> natv; ++iv) {
    pmv_cuv <<< g, b, b.x * sizeof(double) >>>
      (ds, dhuv + (iv * ce -> ngrid), dt + (iv * ce -> ngrid));
    thrust::device_ptr<double> ds_ptr(ds);
    double s = thrust::reduce(ds_ptr, ds_ptr + g.x * g.y);
    cuv += s * sv -> rhov[iv];
  }
  cuv = cuv * ce -> dv;
  double pmv = sv -> xikt * (1.0 - cuv);

  return pmv;
}


__global__ void pmv_cuv(double * ds, double2 * dhuv, double * dt) {
  extern __shared__ double sdata[];

  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;

  sdata[threadIdx.x] = dhuv[ip].x - dt[ip];
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
    if (threadIdx.x < s) {
      sdata[threadIdx.x] += sdata[threadIdx.x + s];
    }
    __syncthreads();
  }
  if (threadIdx.x < 32) {
    volatile double *smem = sdata;
    smem[threadIdx.x] += smem[threadIdx.x + 32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 2];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 1];
  }
  if (threadIdx.x == 0) ds[blockIdx.x + blockIdx.y * gridDim.x] = sdata[0];
}
