#include "hip/hip_runtime.h"
#include <iostream>
#include "rism3d.h"

void RISM3D :: cal_LJ() {
  __global__ void LJ(double * du, const double * __restrict__ dsig, 
		     const double * __restrict__ deps, 
		     const double4 * __restrict__ dru,
		     double cut2, double ikbt, double bx, double by, double bz,
		     int nx, int ny, int nz, int natu, int iv);

  const double cut = 1.0e-2;
  const double cut2 = cut * cut;

  cout << "tabulating solute Lennard-Jones potential ..." << endl;

  hipMalloc(&du, ce -> ngrid * sv -> natv * sizeof(double));
  hipMalloc(&dsig, su -> num * sv -> natv * sizeof(double));
  hipMalloc(&deps, su -> num * sv -> natv * sizeof(double));
  hipMemset(du, 0.0, ce -> ngrid * sv -> natv * sizeof(double));

  siguv = new double[su -> num * sv -> natv];
  epsuv = new double[su -> num * sv -> natv];

  for (int iv = 0; iv < sv -> natv; ++iv) {
#pragma omp parallel for
    for (int iu = 0; iu < su -> num; ++iu) {
      int ip = iu + su -> num * iv;
      siguv[ip] = (su -> sig[iu] + sv -> sigv[iv]) * 0.5;
      epsuv[ip] = sqrt (su -> eps[iu] * sv -> epsv[iv] * kcal2J);
    }
  }

  hipMemcpyAsync(dsig, siguv, su -> num * sv -> natv * sizeof(double),
		  hipMemcpyDefault);
  hipMemcpyAsync(deps, epsuv, su -> num * sv -> natv * sizeof(double),
		  hipMemcpyDefault);

  double iKbT = 1.0 / (avogadoro * boltzmann * sv -> temper);
  for (int iv = 0; iv < sv -> natv; ++iv) {
    LJ <<< g, b >>> (du + (iv * ce -> ngrid), dsig, deps, su -> dr, 
		      cut2, iKbT, ce -> dr[0], ce -> dr[1], ce -> dr[2], 
		      ce -> grid[0], ce -> grid[1], ce -> grid[2], 
		      su -> num, iv);
  }
}

__global__ void LJ(double * du, const double * __restrict__ dsig, 
		   const double * __restrict__ deps, 
		   const double4 * __restrict__ dru,
                   double cut2, double ikbt, double bx, double by, double bz,
                   int nx, int ny, int nz, int natu, int iv) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  double rx = ((int)threadIdx.x - nx / 2) * bx;
  double ry = ((int)blockIdx.x - ny / 2) * by;
  double rz = ((int)blockIdx.y - nz / 2) * bz;
  for (int iu = 0; iu < natu; ++iu) {
    int iuv = iu + natu * iv;
    double dx = rx - dru[iu].x;
    double dy = ry - dru[iu].y;
    double dz = rz - dru[iu].z;
    double r2 = dx * dx + dy * dy + dz * dz ;

    if (r2 < cut2) r2 = cut2;

    double irs2 = dsig[iuv] * dsig[iuv] / r2;

    double irs6 = irs2 * irs2 * irs2;
    du[ip] += deps[iuv] * 4.0 * irs6 * (irs6 - 1.0);
  }
  du[ip] *= ikbt;
}
